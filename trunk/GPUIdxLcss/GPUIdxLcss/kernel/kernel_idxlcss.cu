#include "hip/hip_runtime.h"
#ifndef _LCSS_IDXKERNEL_H_
#define _LCSS_IDXKERNEL_H_
#include <cutil.h>
#include <cutil_inline.h>
__global__ static void Inverted_List_Create(int* T,int c,int* res,int N)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	__shared__ int cnt;
	__syncthreads();
	if(idx == 0)
		cnt = 0;
	__syncthreads();
	if(idx < N)
	{
		if(T[idx] == c)
		{
			res[cnt++] = idx;
		}
	}
}
#endif