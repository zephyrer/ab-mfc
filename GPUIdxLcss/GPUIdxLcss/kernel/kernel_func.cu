#include "hip/hip_runtime.h"
#ifndef _LCSS_KERNELFUN_H_
#define _LCSS_KERNELFUN_H_
#include <cutil.h>
#include <cutil_inline.h>
__global__ static void kernel(float* source,int len,float *result)
{
	int i;
	float sum;
	sum = 0;
	for (i=0;i<len;i++)
		sum+=*(source+i);
	*result = sum;
}

__global__ static void ArrayAdd(float* res,float a,clock_t* time)
{
	clock_t start = clock();
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<32)
	{
		res[idx] = idx + a;
	}
	*time = clock() - start;
}
__global__ static void SumMatrix(float* A,float* B)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	__shared__ int s_data[128];
	s_data[tid]=A[bid*128+tid];
	__syncthreads();
	for (int i=64;i>0;i/=2)
	{
		if(tid<i)
		{
			s_data[tid]=s_data[tid]+s_data[tid+i];
		}
		__syncthreads();
	}
	if (tid==0)
	{
		B[bid]=s_data[0];
	}
}
#endif