/********************************************************************
*  main.cu
*  This is the start point of the project!
*********************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <iostream>
#include <vector>
#include <indexing/lcss.h>
#include <kernel/kernel_func.cu>
#include <kernel/kernel_idxlcss.cu>
/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;
}

#endif

int main(int argc, char* argv[])
{
	float h_A[32*128],h_B[128];
	float *d_A,*d_B;
	unsigned int size = 32*128*sizeof(float);


	if(!InitCUDA()) {
		return 0;
	}

	char	*device_result	= 0;
	char	host_result[12]	={0};

	cutilSafeCall( hipMalloc((void**) &device_result, sizeof(char) * 11));

	unsigned int timer = 0;
	cutilCheckError( cutCreateTimer( &timer));
	cutilCheckError( cutStartTimer( timer));
//////////////////////////////////��������////////////////////////////////////////
	hipMalloc((void**)&d_A,size);
	hipMalloc((void**)&d_B,32*sizeof(float));
	for(int i=0;i<32*128;i++)
	{
		h_A[i] = 1;
	}
	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	SumMatrix<<<32,128>>>(d_A,d_B);
	hipMemcpy(h_B,d_B,32*sizeof(float),hipMemcpyDeviceToHost);
	for(int i=0;i<32;i++)
	{
		std::cout<<h_B[i]<<" ";
	}
	std::cout<<std::endl;
	
	std::vector<int> v1,v2;
//	int n,m,t;
// 	while(std::cin>>n>>m&&n&&m)
// 	{
// 		v1.clear();
// 		v2.clear();
// 		for(int i=0;i<n;i++)
// 		{
// 			std::cin>>t;
// 			v1.push_back(t);
// 		}
// 		for(int i=0;i<m;i++)
// 		{
// 			std::cin>>t;
// 			v2.push_back(t);
// 		}
// 		std::cout<<LCSS(v1,v2)<<std::endl;
// 	}
	int text[8] = {1,2,3,4,1,3,1,4},num;
	int res[8];
	int * d_T,*d_res;
	size = 8*sizeof(int);
	hipMalloc((void**)&d_res,size);
	hipMalloc((void**)&d_T,size);
	hipMemcpy(d_T,text,size,hipMemcpyHostToDevice);
	Inverted_List_Create<<<1,8>>>(d_T,1,d_res,num);
	hipMemcpy(res,d_res,size,hipMemcpyDeviceToHost);
	for(int i=0;i<num;i++)
	{
		std::cout<<res[i]<<" ";
	}
	std::cout<<std::endl;
/////////////////////////////////////////////////////////////////////////////////
	cutilCheckMsg("Kernel execution failed\n");
	hipDeviceSynchronize();
	cutilCheckError( cutStopTimer( timer));
	printf("Processing time: %f (ms)\n", cutGetTimerValue( timer));
	cutilCheckError( cutDeleteTimer( timer));

	cutilSafeCall( hipMemcpy(host_result, device_result, sizeof(char) * 11, hipMemcpyDeviceToHost));
	printf("%s\n", host_result);

	cutilSafeCall( hipFree(device_result));

	return 0;
}
